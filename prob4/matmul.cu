
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <stdio.h>
using namespace std;

#define BLOCK_SIZE 16
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString( err ), file, line);
        exit(EXIT_FAILURE);
    }
}


/*
    Convolution
*/
__global__ void gpu_multABtoC(float *a,float *b, float *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0;
    if( col < k && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
} 

extern "C"{
    void matmul(float *C, float *A, float *B, int m, int n, int k)
    {
        // Allocate memory space on the device 
        float *dev_a, *dev_b, *dev_c;
        hipMalloc((void **) &dev_a, sizeof(float)*m*n);
        hipMalloc((void **) &dev_b, sizeof(float)*n*k);
        hipMalloc((void **) &dev_c, sizeof(float)*m*k);

        // copy matrix A and B from host to device memory
        hipMemcpy(dev_a, A, sizeof(float)*m*n, hipMemcpyHostToDevice);
        hipMemcpy(dev_b, B, sizeof(float)*n*k, hipMemcpyHostToDevice);
        hipMemcpy(dev_c, C, sizeof(float)*m*k, hipMemcpyHostToDevice);

        unsigned int gridev_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
        unsigned int gridev_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
        dim3 dimGrid(gridev_cols, gridev_rows);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    
        // Launch kernel 
        gpu_multABtoC<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, m, n, k);    

        // Transefr results from device to host 
        hipMemcpy(C, dev_c, sizeof(float)*m*k, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        // free memory
        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);
    }
}